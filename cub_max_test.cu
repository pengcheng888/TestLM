#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <cstdlib>

__global__ void find_max_kernel(const float* input, float* output, int size) {
    // 使用共享内存进行块内归约
    typedef hipcub::BlockReduce<float, 256> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    
    float thread_data = -__FLT_MAX__;
    
    // 每个线程处理一个或多个元素
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        thread_data = input[tid];
    }
    
    // 使用 hipcub::Max() 进行块内归约
    float block_max = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Max());
    
    // 第一个线程存储结果
    if (threadIdx.x == 0) {
        output[blockIdx.x] = block_max;
    }
}

int main() {
    const int N = 1024;
    const int block_size = 256;
    const int grid_size = (N + block_size - 1) / block_size;
    
    // 创建主机数据
    float* h_data = new float[N];
    for (int i = 0; i < N; i++) {
        h_data[i] = static_cast<float>(rand()) / RAND_MAX; // 0.0 - 1.0
    }
    
    // 创建设备内存
    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, grid_size * sizeof(float));
    
    // 复制数据到设备
    hipMemcpy(d_input, h_data, N * sizeof(float), hipMemcpyHostToDevice);
    
    // 启动内核
    find_max_kernel<<<grid_size, block_size>>>(d_input, d_output, N);
    
    // 复制结果回主机
    float* h_output = new float[grid_size];
    hipMemcpy(h_output, d_output, grid_size * sizeof(float), hipMemcpyDeviceToHost);
    
    // 在主机上计算最终最大值
    float final_max = -__FLT_MAX__;
    for (int i = 0; i < grid_size; i++) {
        if (h_output[i] > final_max) {
            final_max = h_output[i];
        }
    }
    
    // 在主机上验证
    float host_max = -__FLT_MAX__;
    for (int i = 0; i < N; i++) {
        if (h_data[i] > host_max) {
            host_max = h_data[i];
        }
    }
    
    // 输出结果
    std::cout << "Host computed max: " << host_max << std::endl;
    std::cout << "CUB computed max: " << final_max << std::endl;
    
    // 验证结果
    if (std::abs(host_max - final_max) < 1e-5) {
        std::cout << "\033[32mTest PASSED\033[0m" << std::endl;
    } else {
        std::cout << "\033[31mTest FAILED\033[0m" << std::endl;
    }
    
    // 清理资源
    delete[] h_data;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);
    
    return 0;
} // nvcc cub_max_test.cu -o cub_max_test -I/usr/local/cuda/include -L/usr/local/cuda/lib64 -lcudart
