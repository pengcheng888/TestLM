
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>
#include <iostream>
#include <cstdlib>

int main() {
    hiprtcProgram prog;
    const char *src = "extern \"C\" __global__ void kernel() {}";
    
    // 创建 NVRTC 程序
    hiprtcResult result = hiprtcCreateProgram(&prog, src, "test.cu", 0, NULL, NULL);
    
    if (result != HIPRTC_SUCCESS) {
        std::cerr << "nvrtcCreateProgram failed: " << hiprtcGetErrorString(result) << std::endl;
        return EXIT_FAILURE;
    }
    
    // 编译程序
    const char *opts[] = {"--gpu-architecture=compute_80"};
    result = hiprtcCompileProgram(prog, 1, opts);
    
    if (result != HIPRTC_SUCCESS) {
        size_t logSize;
        hiprtcGetProgramLogSize(prog, &logSize);
        char *log = new char[logSize];
        hiprtcGetProgramLog(prog, log);
        std::cerr << "Compilation failed:\n" << log << std::endl;
        delete[] log;
        return EXIT_FAILURE;
    }
    
    // 清理资源
    hiprtcDestroyProgram(&prog);
    std::cout << "NVRTC test succeeded!" << std::endl;
    return EXIT_SUCCESS;
}
